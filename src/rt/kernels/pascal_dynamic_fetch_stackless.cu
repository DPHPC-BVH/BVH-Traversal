#include "hip/hip_runtime.h"
/*
 *  Copyright (c) 2009-2011, NVIDIA Corporation
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are met:
 *      * Redistributions of source code must retain the above copyright
 *        notice, this list of conditions and the following disclaimer.
 *      * Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimer in the
 *        documentation and/or other materials provided with the distribution.
 *      * Neither the name of NVIDIA Corporation nor the
 *        names of its contributors may be used to endorse or promote products
 *        derived from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 *  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 *  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
 *  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 *  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 *  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 *  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    GK104-optimized variant of the "Persistent speculative
    while-while" kernel used in:

    "Understanding the Efficiency of Ray Traversal on GPUs",
    Timo Aila and Samuli Laine,
    Proc. High-Performance Graphics 2009

    This variant fetches new work dynamically as soon as the
    warp occupancy drops below a pre-determined threshold.
*/

#include "CudaTracerKernels.hpp"

//------------------------------------------------------------------------

#define DEBUG 1

#define DYNAMIC_FETCH_THRESHOLD 20          // If fewer than this active, fetch new rays

extern "C" __device__ int g_warpCounter;    // Work counter for persistent threads.

//------------------------------------------------------------------------

extern "C" __global__ void queryConfig(void)
{
    g_config.bvhLayout = BVHLayout_Compact2_Stackless;
    g_config.blockWidth = 32;
    g_config.blockHeight = 4;
    g_config.usePersistentThreads = 1;
}

//------------------------------------------------------------------------

TRACE_FUNC
{
    // Traversal stack in CUDA thread-local memory.

    // Live state during traversal, stored in registers.

    float   origx, origy, origz;            // Ray origin.
    char*   stackPtr;                       // Current position in traversal stack.
    int     leafAddr;                       // First postponed leaf, non-negative if none.
    int     lastNodeAddr;                   // Parent node
    int     nodeAddr = EntrypointSentinel;  // Non-negative: current internal node, negative: second postponed leaf.
    int     hitIndex;                       // Triangle index of the closest intersection, -1 if none.
    float   hitT;                           // t-value of the closest intersection.
    float   tmin;
    int     rayidx;
    float   oodx;
    float   oody;
    float   oodz;
    float   dirx;
    float   diry;
    float   dirz;
    float   idirx;
    float   idiry;
    float   idirz;

    // Initialize persistent threads.

    __shared__ volatile int nextRayArray[MaxBlockHeight]; // Current ray index in global buffer.

    // Persistent threads: fetch and process rays in a loop.

    do
    {
        const int tidx = threadIdx.x;
        volatile int& rayBase = nextRayArray[threadIdx.y];

        // Fetch new rays from the global pool using lane 0.

        const bool          terminated     = nodeAddr==EntrypointSentinel;
        const unsigned int  maskTerminated = __ballot(terminated);
        const int           numTerminated  = __popc(maskTerminated);
        const int           idxTerminated  = __popc(maskTerminated & ((1u<<tidx)-1));

        #ifdef DEBUG1
            float4 croot=FETCH_TEXTURE(nodesA, 3, float4); // (c0, c1, p, dim)
            int root_parent = __float_as_int(croot.z);
            printf("Root Parent: %x\n", root_parent);
        #endif

        if(terminated)
        {
            if (idxTerminated == 0)
                rayBase = atomicAdd(&g_warpCounter, numTerminated);

            rayidx = rayBase + idxTerminated;
            if (rayidx >= numRays)
                break;

            // Fetch ray.

            float4 o = FETCH_GLOBAL(rays, rayidx * 2 + 0, float4);
            float4 d = FETCH_GLOBAL(rays, rayidx * 2 + 1, float4);
            origx = o.x;
            origy = o.y;
            origz = o.z;
            tmin  = o.w;
            dirx  = d.x;
            diry  = d.y;
            dirz  = d.z;
            hitT  = d.w;
            float ooeps = exp2f(-80.0f); // Avoid div by zero.
            idirx = 1.0f / (fabsf(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
            idiry = 1.0f / (fabsf(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
            idirz = 1.0f / (fabsf(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
            oodx  = origx * idirx;
            oody  = origy * idiry;
            oodz  = origz * idirz;

            // Setup traversal.

            leafAddr = 0;   // No postponed leaf.
            nodeAddr = 0;   // Start from the root.
            lastNodeAddr = EntrypointSentinel;
            hitIndex = -1;  // No triangle intersected so far.
        }

        // Traversal loop.

        while(nodeAddr != EntrypointSentinel)
        {
            int current_child;
            #ifdef DEBUG1
                printf("Entered Main loop iteration: %i\n", nodeAddr);
            #endif
            // Traverse internal nodes until all SIMD lanes have found a leaf.

//          while (nodeAddr >= 0 && nodeAddr != EntrypointSentinel)
            while (unsigned int(nodeAddr) < unsigned int(EntrypointSentinel))   // functionally equivalent, but faster
            {
                // Fetch AABBs of the two child nodes.

                #ifdef DEBUG1
                    if(nodeAddr != 0){
                        printf("Entered Main loop iteration (not root): %i\n", nodeAddr);
                    }
                    
                #endif

                float4 cnodes=FETCH_TEXTURE(nodesA, nodeAddr+3, float4); // (c0, c1, p, dim)
                int nearChild = __float_as_int(cnodes.x);
                int farChild = __float_as_int(cnodes.y);
                int parent = __float_as_int(cnodes.z);
                int nch_idx = 0;
                int fch_idx = 1;

                // get near and far child
                int dim = __float_as_int(cnodes.w);

                float ray_dim = 0.0f;
                switch(dim){
                    case 0: 
                        ray_dim = idirx;
                        break;
                    case 1:
                        ray_dim = idiry;
                        break;
                case 2:
                        ray_dim = idirz;
                        break;
                }

                if(ray_dim < 0.0f){
                    swap(nearChild, farChild);
                    swap(nch_idx, fch_idx);
                }

                if(lastNodeAddr == farChild){
                    lastNodeAddr = nodeAddr;
                    nodeAddr = parent;
                    continue;
                }

                // if we come from parent -> nearChild, if we come from sibling -> farChild
                current_child = (lastNodeAddr == parent) ? nearChild : farChild;

                // 0 if currentChild is c0, 1 if currentchild is c1
                const int current_child_idx = (lastNodeAddr == parent) ? nch_idx : fch_idx;

                
                const float4 nxy = FETCH_TEXTURE(nodesA, nodeAddr+current_child_idx, float4);  // (c0/1.lo.x, c0/1.hi.x, c0/1.lo.y, c0/1.hi.y)
                const float4 nz   = FETCH_TEXTURE(nodesA, nodeAddr+2, float4);  // (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)

                const float c0lox = nxy.x * idirx - oodx;
                const float c0hix = nxy.y * idirx - oodx;
                const float c0loy = nxy.z * idiry - oody;
                const float c0hiy = nxy.w * idiry - oody;

                float nz_x;
                float nz_y;

                // TODO: replace branch with pointer arith.
                if(current_child_idx == 0){
                    nz_x = nz.x;
                    nz_y = nz.y;
                }else{
                    nz_x = nz.z;
                    nz_y = nz.w;
                }
                
                const float c0loz = nz_x * idirz - oodz;
                const float c0hiz = nz_y * idirz - oodz;
                const float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
                const float c0max = spanEndKepler  (c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
                //float c0min = max4(fminf(c0lox, c0hix), fminf(c0loy, c0hiy), fminf(c0loz, c0hiz), aux->tmin);
                //float c0max = min4(fmaxf(c0lox, c0hix), fmaxf(c0loy, c0hiy), fmaxf(c0loz, c0hiz), hitT);
                const int traverseCurrentChild = (c0max >= c0min);

                if(traverseCurrentChild){
                    // if we hit the BB -> go down a level
                    lastNodeAddr = nodeAddr;
                    nodeAddr = current_child;
                     #ifdef DEBUG1
                        printf("Found intersection: %i\n", nodeAddr);
                    #endif
               
                }else{
        
                    // otherwise:
                    // if we are nearChild  -> go to far child
                    // else                 -> go up a level
                    if(current_child == nearChild){
                        lastNodeAddr = nearChild;
                    }else{
                        lastNodeAddr = nodeAddr;
                        nodeAddr = parent;
                    }
                
                    continue;
                }

                // c
                /*
                if (nodeAddr < 0 && leafAddr >= 0){
                    #ifdef DEBUG1
                        printf("Found leaf: %i\n", nodeAddr);
                    #endif
                    leafAddr = nodeAddr;
                    nodeAddr = lastNodeAddr;
                    lastNodeAddr = current_child;
                }
                */

                // All SIMD lanes have found a leaf? => process them.

                // NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;".
                // tried everything with CUDA 4.2 but always got several redundant instructions.
                /*
                unsigned int mask;
                asm("{\n"
                    "   .reg .pred p;               \n"
                    "setp.ge.s32        p, %1, 0;   \n"
                    "vote.ballot.b32    %0,p;       \n"
                    "}"
                    : "=r"(mask)
                    : "r"(leafAddr));
                if(!mask){
                    #ifdef DEBUG1
                        printf("Warp got out: %i\n", nodeAddr);
                    #endif
                    break;
                }
                */
                    

                //if(!__any(leafAddr >= 0))
                //    break;
            }

            // Process postponed leaf nodes.

            while (leafAddr < 0)
            {
                nodeAddr = lastNodeAddr;
                lastNodeAddr = current_child;
                for (int triAddr = ~leafAddr;; triAddr += 3)
                {
                    // Tris in TEX (good to fetch as a single batch)
                    const float4 v00 = tex1Dfetch(t_trisA, triAddr + 0);
                    const float4 v11 = tex1Dfetch(t_trisA, triAddr + 1);
                    const float4 v22 = tex1Dfetch(t_trisA, triAddr + 2);

                    // End marker (negative zero) => all triangles processed.
                    if (__float_as_int(v00.x) == 0x80000000)
                        break;

                    float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
                    float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
                    float t = Oz * invDz;

                    if (t > tmin && t < hitT)
                    {
                        // Compute and check barycentric u.

                        float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
                        float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
                        float u = Ox + t*Dx;

                        if (u >= 0.0f)
                        {
                            // Compute and check barycentric v.

                            float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
                            float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
                            float v = Oy + t*Dy;

                            if (v >= 0.0f && u + v <= 1.0f)
                            {
                                // Record intersection.
                                // Closest intersection not required => terminate.

                                hitT = t;
                                hitIndex = triAddr;
                                if (anyHit)
                                {
                                    nodeAddr = EntrypointSentinel;
                                    break;
                                }
                            }
                        }
                    }
                } // triangle
                // finished leaf
                //leafAddr = 0;

                // Another leaf was postponed => process it as well.

//              if(leafAddr2<0) { leafAddr = leafAddr2; leafAddr2=0; } else     // postpone2
                /*{
                    leafAddr = nodeAddr;
                    if (nodeAddr < 0)
                    {
                        nodeAddr = *(int*)stackPtr;
                        stackPtr -= 4;
                    }
                }*/
            } // leaf

            // DYNAMIC FETCH

            if( __popc(__ballot(true)) < DYNAMIC_FETCH_THRESHOLD )
                break;

        } // traversal

        // Remap intersected triangle index, and store the result.

        if (hitIndex == -1) { STORE_RESULT(rayidx, -1, hitT); }
        else                { STORE_RESULT(rayidx, FETCH_TEXTURE(triIndices, hitIndex, int), hitT); }

    } while(true);
}

//------------------------------------------------------------------------
